#include "hip/hip_runtime.h"
#define CPLUSPLUS

#include <stdio.h>
#include "MDSystem_interface.h"
#include "common.h"
#include "BoxGeometry.h"
#include "MDSystem.h"
#include "RandomGenerator.h"
#include "Auxiliary.h"
#include "NeighborList_interface.h"
#include"Statistic.h"
#include "Integrator_interface.h"
#include "InteractionEngine_interface.h"
#include "tmp.h"
#include "Reshuffle_interface.h"
#include "Displacement_interface.h"

#include "Topology.h"
#include "SystemBondedInteraction.h"

#include "BondInteraction.h"
#include "NonBondedInteraction.h"

#include "xdrfile/xdrfile.h"
#include "xdrfile/xdrfile_xtc.h"
// #include "GroFileManager.h"
#include "ErrorProfile.h"
#include "AssignForceCorr.h"

#define NThreadsPerBlockCell	128
#define NThreadsPerBlockAtom	96

int main(int argc, char * argv[])
{
  char * filename;
  char xtcname[] = "traj.xtc";
  char afcname[] = "fc.ctj";

  if (argc != 7){
    printf ("Usage:\n%s conf.gro device refh rcut1 rcut2 start_t\n", argv[0]);
    return 1;
  }
  filename = argv[1];
  printf ("# setting device to %d\n", atoi(argv[2]));
  hipSetDevice (atoi(argv[2]));
  checkCUDAError ("set device");
  double refh = atof (argv[3]);
  ScalorType start_t = atof (argv[6]);
  
  MDSystem sys;
  sys.initConfig(filename);

  Topology::System sysTop;
  Topology::Molecule mol;
  mol.pushAtom (Topology::Atom (1.0, 0.0, 0));
  LennardJones6_12Parameter ljparam;
  ScalorType rcut1 = atof(argv[4]);
  ScalorType rcut2 = atof(argv[5]);
  printf ("# rcut2 is %f\n", rcut2);
  int nimage = (rcut2 - 0.00001) / sys.box.size.y;
  nimage ++;
  printf ("#@ nimage is %d\n", nimage);

  ljparam.reinit (1.f, 1.f, 0.f, rcut1, rcut2);
  
  sysTop.addNonBondedInteraction (Topology::NonBondedInteraction(0, 0, ljparam));
  sysTop.addMolecules (mol, sys.hdata.numAtom);

  sys.initTopology (sysTop);
  sys.initDeviceData ();
  
  SystemNonBondedInteraction sysNbInter;
  sysNbInter.reinit (sysTop);
  
  InteractionEngine inter (sys, NThreadsPerBlockAtom);
  inter.registNonBondedInteraction (sysNbInter);

  AssignForceCorr afc;
  afc.reinit (sys, NThreadsPerBlockAtom);
  afc.init_read (afcname);
  
  int step;
  int natoms= 0;
  float time, afctime, prec;
  matrix gbox;  
  rvec * xx;
  xx = (rvec *) malloc (sizeof(rvec) * sys.hdata.numAtom);
  XDRFILE * fpxtc = xdrfile_open (xtcname, "r");
  if (fpxtc == NULL){
    fprintf (stderr, "cannot open file %s\n", xtcname);
    return 1;;
  }
  std::vector<double > boxsize (3);
  boxsize[0] = sys.box.size.x;
  boxsize[1] = sys.box.size.y;
  boxsize[2] = sys.box.size.z;
  ErrorProfile_PiecewiseConst ep (boxsize, refh);
  std::vector<std::vector<double > > coord, force;
  coord.resize (sys.hdata.numAtom, std::vector<double > (3, 0.));
  force.resize (sys.hdata.numAtom, std::vector<double > (3, 0.));
  while (read_xtc (fpxtc, natoms, &step, &time, gbox, xx, &prec) == 0){
    afc.read (afctime);
    if (fabs (time - afctime) > 1e-4) {
      printf ("inconsistent trajactories\n");
      exit (1);
    }
    if (time < start_t - 1e-4) continue;
    printf ("loaded frame at time %f ps       \r", time);
    fflush (stdout);
    for (unsigned i = 0; i < sys.hdata.numAtom; ++i){
      sys.hdata.coord[i].x = xx[i][0];
      sys.hdata.coord[i].y = xx[i][1];
      sys.hdata.coord[i].z = xx[i][2];
    }
    cpyHostMDDataToDevice (&sys.hdata, &sys.ddata);
    inter.clearInteraction (sys);
    inter.applyNonBondedInteraction (sys, rcut2, NULL);
    cpyDeviceMDDataToHost (&sys.ddata, &sys.hdata);
    for (unsigned i = 0; i < sys.hdata.numAtom; ++i){
      coord[i][0] = sys.hdata.coord[i].x;
      coord[i][1] = sys.hdata.coord[i].y;
      coord[i][2] = sys.hdata.coord[i].z;
      ScalorType cfx, cfy, cfz;
      afc.getForceCorr (coord[i][0], coord[i][1], coord[i][2],
			cfx, cfy, cfz);
      force[i][0] = sys.hdata.forcx[i] - cfx;
      force[i][1] = sys.hdata.forcy[i] - cfy;
      force[i][2] = sys.hdata.forcz[i] - cfz;
    }
    ep.deposit (coord, force);
  }
  
  ep.calculate();
  // ep.print_x (("real.x.out"));
  ep.print_x_avg (("a.real.x.out"));
  // ep.print_xy (("real.xy.out"));
  
  return 0;
}

  
