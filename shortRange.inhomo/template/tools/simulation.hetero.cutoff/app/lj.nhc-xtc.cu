#include <stdio.h>
#include "MDSystem_interface.h"
#include "common.h"
#include "BoxGeometry.h"
#include "MDSystem.h"
#include "RandomGenerator.h"
#include "Auxiliary.h"
#include "NeighborList_interface.h"
#include"Statistic.h"
#include "Integrator_interface.h"
#include "InteractionEngine_interface.h"
#include "tmp.h"
#include "Reshuffle_interface.h"
#include "Displacement_interface.h"
#include "AssignRCut.h"

#include "Topology.h"
#include "SystemBondedInteraction.h"

#include "BondInteraction.h"
#include "NonBondedInteraction.h"
#include "PressureCorrection.h"

#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <fftw3.h>

// #define NThreadsPerBlockCell	32
// #define NThreadsPerBlockAtom	4

#define NThreadsPerBlockCell	96
#define NThreadsPerBlockAtom	96

#include "DensityProfile.h"

int main(int argc, char * argv[])
{
  IndexType nstep = 3000000;
  IndexType confFeq = 2000;
  IndexType thermoFeq = 100;
  ScalorType dt = 0.005;
  ScalorType rcut = 7.5;
  ScalorType nlistExten = 0.5;
  ScalorType refT = 1.20;
  ScalorType tauT = 1.0;
  char * filename;

  IndexType densityProfileSamplingFeq = 40;
  IndexType rcutAssignFeq = 40;
  IndexType rcutUpdateFeq = 1000;
  double refh = 1.0;
  double rcmin = 03.0;
  double rcmax = 10.0;
  double rcstep = 0.5;
  double targetPrec = 0.004;
  
  if (argc != 4){
    printf ("Usage:\n%s conf.gro nstep device\n", argv[0]);
    return 1;
  }
  if (argc != 1){
    nstep = atoi(argv[2]);
    filename = argv[1];
  }
  printf ("# setting device to %d\n", atoi(argv[3]));
  hipSetDevice (atoi(argv[3]));
  checkCUDAError ("set device");

  MDSystem sys;
  sys.initConfig(filename);

  Topology::System sysTop;
  Topology::Molecule mol;
  mol.pushAtom (Topology::Atom (1.0, 0.0, 0));
  LennardJones6_12Parameter ljparam;
  ljparam.reinit (1.f, 1.f, 0.f, rcut);
  sysTop.addNonBondedInteraction (Topology::NonBondedInteraction(0, 0, ljparam));
  sysTop.addMolecules (mol, sys.hdata.numAtom);

  sys.initTopology (sysTop);
  sys.initDeviceData ();

  DensityProfile_PiecewiseConst dp;
  printf ("# init DensityProfile_PiecewiseConst\n");
  dp.reinit (sys.box.size.x, sys.box.size.y, sys.box.size.z, refh);
  AdaptRCut arc;
  printf ("# init AdaptRCut\n");
  arc.reinit (rcmin, rcmax, rcstep, dp);
  AssignRCut assign_rcut;
  printf ("# init AssignRCut\n");
  assign_rcut.reinit (sys, arc, NThreadsPerBlockAtom);
  assign_rcut.uniform (rcut);
  assign_rcut.print_x ("rcut.x.out");
  assign_rcut.assign (sys);
  PressureCorrection pc (arc, dp);
  ScalorType pcxx, pcyy, pczz;
  pcxx = pcyy = pczz = 0.;
  
  SystemNonBondedInteraction sysNbInter;
  sysNbInter.reinit (sysTop);
  ScalorType energyCorr = sysNbInter.energyCorrection ();
  ScalorType pressureCorr = sysNbInter.pressureCorrection ();
  
  ScalorType maxrcut = sysNbInter.maxRcut();
  ScalorType rlist = maxrcut + nlistExten;
  CellList clist (sys, rlist, NThreadsPerBlockCell, NThreadsPerBlockAtom);
  CellList clist_resh (sys, 3., NThreadsPerBlockCell, NThreadsPerBlockAtom);
  NeighborList nlist (sysNbInter, sys, rlist, nlistExten, NThreadsPerBlockAtom, 4.f);
  sys.normalizeDeviceData ();
  clist.rebuild (sys, NULL);
  clist_resh.rebuild (sys, NULL);
  nlist.rebuild (sys, clist, NULL);
  Displacement_max disp (sys, NThreadsPerBlockAtom);
  disp.recordCoord (sys);
  
  MDStatistic st(sys);
  TranslationalFreedomRemover tfremover (sys, NThreadsPerBlockAtom);
  InteractionEngine inter (sys, NThreadsPerBlockAtom);
  inter.registNonBondedInteraction (sysNbInter);
  
  MDTimer timer;
  unsigned i;
  ScalorType seed = 1;
  RandomGenerator_MT19937::init_genrand (seed);

  VelocityVerlet inte_vv (sys, NThreadsPerBlockAtom);
  VelocityRescale inte_vr (sys, NThreadsPerBlockAtom, refT, 0.1);
  NoseHoover_Chains2 nhc;
  nhc.reinit (sys, NThreadsPerBlockAtom, refT, tauT);

  Reshuffle resh (sys);
  
  timer.tic(mdTimeTotal);
  if (resh.calIndexTable (clist_resh, &timer)){
    sys.reshuffle   (resh.indexTable, sys.hdata.numAtom, &timer);
    clist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
    clist_resh.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
    nlist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
    disp.reshuffle  (resh.indexTable, sys.hdata.numAtom, &timer);  
  }
  
  printf ("# prepare ok, start to run\n");
  sys.recoverDeviceData (&timer);
  sys.updateHostFromRecovered (&timer);
  sys.writeHostDataGro ("confstart.gro", 0, 0.f, &timer);
  assign_rcut.init_write ("rcut.rtj");
  
  printf ("# prepare ok, start to run\n");
  printf ("#*     1     2           3         4            5       6                7          8          9         10        11   12   13 14\n");
  printf ("#* nstep  time  nonBondedE  kineticE  temperature  totalE  NHC_Hamiltonian pressureXX pressureYY pressureZZ s_tension pcxx pcyy tc\n");

  try{
    sys.initWriteXtc ("traj.xtc");
    sys.recoverDeviceData (&timer);
    sys.updateHostFromRecovered (&timer);
    sys.writeHostDataXtc (0, 0*dt, &timer);
    for (i = 0; i < nstep; ++i){
      if (i%10 == 0){
	tfremover.remove (sys, &timer);
      }
      
      nhc.operator_L (0.5 * dt, sys, &timer);
      inte_vv.step1 (sys, dt, &timer);

      st.clearDevice();
      inter.clearInteraction (sys);
      ScalorType maxdr = disp.calMaxDisplacemant (sys, &timer);
      if (maxdr > nlistExten * 0.5){
	// printf ("# Rebuild at step %09i ... ", i+1);
	// fflush(stdout);
	// rebuild
	sys.normalizeDeviceData (&timer);
	disp.recordCoord (sys);
	clist.rebuild (sys, &timer);
	clist_resh.rebuild (sys, &timer);
	nlist.rebuild (sys, clist, &timer);
	// printf ("done\n");
	// fflush(stdout);
      }
      inter.applyNonBondedInteraction (sys, nlist, st, NULL, &timer);

      if ((i+2) % rcutAssignFeq == 0){
	timer.tic (mdTimeAdaptRCut);
        assign_rcut.assign (sys);
	timer.toc (mdTimeAdaptRCut);
      }
      
      inte_vv.step2 (sys, dt, &timer);
      if ((i+1) % thermoFeq == 0){	
	nhc.operator_L (0.5 * dt, sys, st, &timer);
      }
      else {
	nhc.operator_L (0.5 * dt, sys, &timer);	
      }      

      if ((i+1) % thermoFeq == 0){
	timer.tic (mdTimeDataIO);
	st.updateHost ();
	ScalorType px = st.pressureXX (sys.box);
	ScalorType py = st.pressureYY (sys.box);
	ScalorType pz = st.pressureZZ (sys.box);
	printf ("%09d %05e %.5e %.5e %.5e %.5e %.5e %.5e %.5e %.5e %.5e %.5e %.5e %.5e %.2e\n",
		(i+1),  
		(i+1) * dt, 
		st.nonBondedEnergy(),
		st.kineticEnergy(),
		st.kineticEnergy() * 2. / 3. / (double (sys.hdata.numAtom) - 3.),
		st.nonBondedEnergy() +
		st.kineticEnergy(),
		st.nonBondedEnergy() +
		st.kineticEnergy() +
		nhc.HamiltonianContribution (),
		px, py, pz,
		(px - (py + pz) * 0.5) * sys.box.size.x * 0.5,
		pcxx,
		pcyy,
		(pcxx - (pcyy + pczz) * 0.5) * sys.box.size.x * 0.5,
		double (nlist.calSumNeighbor ())
	    );
	fflush(stdout);
	timer.toc (mdTimeDataIO);
      }

      if ((i+1) % densityProfileSamplingFeq == 0) {
	timer.tic (mdTimeDensityProfile);
	sys.updateHostFromDevice (NULL);
	dp.deposite (sys.hdata.coord, sys.hdata.numAtom);
	timer.toc (mdTimeDensityProfile);
      }

      if ((i+1) % rcutUpdateFeq == 0) {
	// printf ("# update rcut\n");
	timer.tic (mdTimeDensityProfile);
	dp.calculate ();
	dp.print_x ("density.x.out");
	timer.toc (mdTimeDensityProfile);
	timer.tic (mdTimeAdaptRCut);
	arc.calError (dp);
	arc.calRCut (targetPrec);
	arc.print_x ("error.x.out");
	assign_rcut.getRCut (arc);
	assign_rcut.print_x ("rcut.x.out");
	pc.correction (arc, dp);
	pcxx = pc.pxx;
	pcyy = pc.pyy;
	pczz = pc.pzz;
	timer.toc (mdTimeAdaptRCut);
	if (i != nstep - 1) dp.clearData ();
      }
      
      if ((i+1) % confFeq == 0){
      	// printf ("write conf\n");
      	sys.recoverDeviceData (&timer);
      	sys.updateHostFromRecovered (&timer);
      	sys.writeHostDataXtc (i+1, (i+1)*dt, &timer);
	assign_rcut.write ((i+1) * dt);
      }
      
      if ((i+1) % 100 == 0){
      	if (resh.calIndexTable (clist_resh, &timer)){
      	  sys.reshuffle   (resh.indexTable, sys.hdata.numAtom, &timer);
      	  clist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
      	  clist_resh.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
      	  nlist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
      	  disp.reshuffle  (resh.indexTable, sys.hdata.numAtom, &timer);  
      	}
      }
    }
    sys.endWriteXtc();
    sys.recoverDeviceData (&timer);
    sys.updateHostFromRecovered (&timer);
    sys.writeHostDataGro ("confout.gro", nstep, nstep*dt, &timer);
    timer.toc(mdTimeTotal);
    timer.printRecord (stderr);
  }
  catch (MDExcptCuda & e){
    // resh.recoverMDDataToHost (sys, &timer);
    // sys.writeHostDataXtc (i+1, (i+1)*dt, &timer);
    timer.toc(mdTimeTotal);
    timer.printRecord (stderr);
    return 1;
  }
  catch (MDException &e){
    fprintf (stderr, "%s\n", e.what());
    return 1;
  }

  assign_rcut.end_write();
  dp.save ("density.save");
  arc.save_rc ("rcut.save");
  
  return 0;
}

  
