#define CPLUSPLUS

#include <stdio.h>
#include "MDSystem_interface.h"
#include "common.h"
#include "BoxGeometry.h"
#include "MDSystem.h"
#include "RandomGenerator.h"
#include "Auxiliary.h"
#include "NeighborList_interface.h"
#include"Statistic.h"
#include "Integrator_interface.h"
#include "InteractionEngine_interface.h"
#include "tmp.h"
#include "Reshuffle_interface.h"
#include "Displacement_interface.h"

#include "Topology.h"
#include "SystemBondedInteraction.h"

#include "BondInteraction.h"
#include "NonBondedInteraction.h"

#include "xdrfile/xdrfile.h"
#include "xdrfile/xdrfile_xtc.h"
// #include "GroFileManager.h"
#include "ErrorProfile.h"
#include "AdaptRCut.h"
#include "AssignRCut.h"

#define NThreadsPerBlockCell	128
#define NThreadsPerBlockAtom	96

int main(int argc, char * argv[])
{
  char * filename;
  char xtcname[] = "traj.xtc";

  if (argc != 7){
    printf ("Usage:\n%s conf.gro device refh rcut1 rcut2 start_t\n", argv[0]);
    return 1;
  }
  filename = argv[1];
  printf ("# setting device to %d\n", atoi(argv[2]));
  hipSetDevice (atoi(argv[2]));
  checkCUDAError ("set device");
  double refh = atof (argv[3]);
  ScalorType start_t = atof (argv[6]);
  
  MDSystem sys;
  sys.initConfig(filename);

  Topology::System sysTop;
  Topology::Molecule mol;
  mol.pushAtom (Topology::Atom (1.0, 0.0, 0));
  LennardJones6_12Parameter ljparam;
  ScalorType rcut1 = atof(argv[4]);
  ScalorType rcut2 = atof(argv[5]);
  printf ("# rcut2 is %f\n", rcut2);
  int nimage = (rcut2 - 0.00001) / sys.box.size.y;
  nimage ++;
  printf ("#@ nimage is %d\n", nimage);

  ljparam.reinit (1.f, 1.f, 0.f, rcut1, rcut2);
  
  sysTop.addNonBondedInteraction (Topology::NonBondedInteraction(0, 0, ljparam));
  sysTop.addMolecules (mol, sys.hdata.numAtom);

  sys.initTopology (sysTop);
  sys.initDeviceData ();
  
  SystemNonBondedInteraction sysNbInter;
  sysNbInter.reinit (sysTop);
  
  InteractionEngine inter (sys, NThreadsPerBlockAtom);
  inter.registNonBondedInteraction (sysNbInter);

  int step;
  int natoms= 0;
  float time, prec;
  matrix gbox;  
  rvec * xx;
  xx = (rvec *) malloc (sizeof(rvec) * sys.hdata.numAtom);
  XDRFILE * fpxtc = xdrfile_open (xtcname, "r");
  if (fpxtc == NULL){
    fprintf (stderr, "cannot open file %s\n", xtcname);
    return 1;;
  }
  std::vector<double > boxsize (3);
  boxsize[0] = sys.box.size.x;
  boxsize[1] = sys.box.size.y;
  boxsize[2] = sys.box.size.z;
  ErrorProfile_PiecewiseConst ep (boxsize, refh);
  std::vector<std::vector<double > > coord, force;
  coord.resize (sys.hdata.numAtom, std::vector<double > (3, 0.));
  force.resize (sys.hdata.numAtom, std::vector<double > (3, 0.));
  while (read_xtc (fpxtc, natoms, &step, &time, gbox, xx, &prec) == 0){
    if (time < start_t - 1e-4) continue;
    printf ("loaded frame at time %f ps       \r", time);
    fflush (stdout);
    for (unsigned i = 0; i < sys.hdata.numAtom; ++i){
      sys.hdata.coord[i].x = xx[i][0];
      sys.hdata.coord[i].y = xx[i][1];
      sys.hdata.coord[i].z = xx[i][2];
    }
    cpyHostMDDataToDevice (&sys.hdata, &sys.ddata);
    inter.clearInteraction (sys);
    inter.applyNonBondedInteraction (sys, rcut2, NULL);
    cpyDeviceMDDataToHost (&sys.ddata, &sys.hdata);
    for (unsigned i = 0; i < sys.hdata.numAtom; ++i){
      coord[i][0] = sys.hdata.coord[i].x;
      coord[i][1] = sys.hdata.coord[i].y;
      coord[i][2] = sys.hdata.coord[i].z;
      force[i][0] = sys.hdata.forcx[i];
      force[i][1] = sys.hdata.forcy[i];
      force[i][2] = sys.hdata.forcz[i];
    }
    ep.deposit (coord, force);
  }
  
  ep.calculate();
  // ep.print_x (("real.x.out"));
  ep.print_x_avg (("a.real.x.out"));
  // ep.print_xy (("real.xy.out"));
  
  return 0;
}

  
