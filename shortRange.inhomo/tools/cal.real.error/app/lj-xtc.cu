#define CPLUSPLUS

#include <stdio.h>
#include "MDSystem_interface.h"
#include "common.h"
#include "BoxGeometry.h"
#include "MDSystem.h"
#include "RandomGenerator.h"
#include "Auxiliary.h"
#include "NeighborList_interface.h"
#include"Statistic.h"
#include "Integrator_interface.h"
#include "InteractionEngine_interface.h"
#include "tmp.h"
#include "Reshuffle_interface.h"
#include "Displacement_interface.h"

#include "Topology.h"
#include "SystemBondedInteraction.h"

#include "BondInteraction.h"
#include "NonBondedInteraction.h"

#include "xdrfile/xdrfile.h"
#include "xdrfile/xdrfile_xtc.h"
// #include "GroFileManager.h"
#include "ErrorProfile.h"

#define NThreadsPerBlockCell	128
#define NThreadsPerBlockAtom	96

int main(int argc, char * argv[])
{
  ScalorType rcut1 = 5.f;
  char * filename;
  char * xtcname;

  if (argc != 6){
    printf ("Usage:\n%s conf.gro traj.xtc rcut1 device refh\n", argv[0]);
    return 1;
  }
  if (argc != 1){
    rcut1 = atof(argv[3]);
    filename = argv[1];
    xtcname  = argv[2];
  }
  printf ("# setting device to %d\n", atoi(argv[4]));
  hipSetDevice (atoi(argv[4]));
  checkCUDAError ("set device");
  double refh = atof (argv[5]);

  MDSystem sys;
  sys.initConfig(filename);

  Topology::System sysTop;
  Topology::Molecule mol;
  mol.pushAtom (Topology::Atom (1.0, 0.0, 0));
  LennardJones6_12Parameter ljparam;
  // ScalorType rcut2 = sys.box.size.z / 2 - 1.f;
  ScalorType rcut2 = rcut1 * 3;
  if (rcut2 > sys.box.size.z / 2.f - 1.f) rcut2 = sys.box.size.z / 2 - 1.f;
  printf ("# rcut1 is %f\n", rcut1);
  printf ("# rcut2 is %f\n", rcut2);
  ljparam.reinit (1.f, 1.f, 0.f, rcut1, rcut2);
  
  sysTop.addNonBondedInteraction (Topology::NonBondedInteraction(0, 0, ljparam));
  sysTop.addMolecules (mol, sys.hdata.numAtom);

  sys.initTopology (sysTop);
  sys.initDeviceData ();
  
  SystemNonBondedInteraction sysNbInter;
  sysNbInter.reinit (sysTop);
  
  // ScalorType maxrcut = sysNbInter.maxRcut();
  // ScalorType rlist = rcut2;
  // CellList clist (sys, rlist, NThreadsPerBlockCell, NThreadsPerBlockAtom);
  // NeighborList nlist (sysNbInter, sys, rlist, NThreadsPerBlockAtom, 2.f);
  // sys.normalizeDeviceData ();
  // clist.rebuild (sys, NULL);
  // nlist.rebuild (sys, clist, NULL);
  InteractionEngine inter (sys, NThreadsPerBlockAtom);
  inter.registNonBondedInteraction (sysNbInter);
  

  inter.clearInteraction (sys);
  inter.applyNonBondedInteraction (sys, rcut2);
    
  sys.updateHostFromDevice (NULL);
  FILE *fp = fopen ("force.out", "w");
  fprintf (fp, "%d\n%f %f %f\n",
	   sys.ddata.numAtom,
	   sys.box.size.x, sys.box.size.y, sys.box.size.z);
  for (unsigned i = 0; i < sys.ddata.numAtom; ++i){
    fprintf (fp, "%e %e %e  %e %e %e\n",
	     sys.hdata.coord[i].x, 
	     sys.hdata.coord[i].y, 
	     sys.hdata.coord[i].z,
	     sys.hdata.forcx[i],
	     sys.hdata.forcy[i],
	     sys.hdata.forcz[i]);
  }
  fclose (fp);

  
  
  int step;
  int natoms= 0;
  float time, prec;
  matrix gbox;  
  rvec * xx;
  xx = (rvec *) malloc (sizeof(rvec) * sys.hdata.numAtom);
  XDRFILE * fpxtc = xdrfile_open (xtcname, "r");
  if (fpxtc == NULL){
    fprintf (stderr, "cannot open file %s\n", xtcname);
    return 1;;
  }
  std::vector<double > boxsize (3);
  boxsize[0] = sys.box.size.x;
  boxsize[1] = sys.box.size.y;
  boxsize[2] = sys.box.size.z;
  ErrorProfile_PiecewiseConst ep (boxsize, refh);
  std::vector<std::vector<double > > coord, force;
  coord.resize (sys.hdata.numAtom, std::vector<double > (3, 0.));
  force.resize (sys.hdata.numAtom, std::vector<double > (3, 0.));
  while (read_xtc (fpxtc, natoms, &step, &time, gbox, xx, &prec) == 0){
    printf ("loaded frame at time %f ps       \r", time);
    fflush (stdout);
    for (unsigned i = 0; i < sys.hdata.numAtom; ++i){
      sys.hdata.coord[i].x = xx[i][0];
      sys.hdata.coord[i].y = xx[i][1];
      sys.hdata.coord[i].z = xx[i][2];
    }
    cpyHostMDDataToDevice (&sys.hdata, &sys.ddata);
    inter.clearInteraction (sys);
    inter.applyNonBondedInteraction (sys, rcut2);
    cpyDeviceMDDataToHost (&sys.ddata, &sys.hdata);

    for (unsigned i = 0; i < sys.hdata.numAtom; ++i){
      coord[i][0] = sys.hdata.coord[i].x;
      coord[i][1] = sys.hdata.coord[i].y;
      coord[i][2] = sys.hdata.coord[i].z;
      force[i][0] = sys.hdata.forcx[i];
      force[i][1] = sys.hdata.forcy[i];
      force[i][2] = sys.hdata.forcz[i];
    }
    ep.deposit (coord, force);
  }
  ep.calculate();
  ep.print_x (("real.x.out"));
  ep.print_xy (("real.xy.out"));
  xdrfile_close (fpxtc);
  free (xx);
  
  return 0;
}

  
